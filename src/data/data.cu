
#include <hip/hip_runtime.h>
/*!
 * Copyright 2018 by xgboost contributors
 */

#ifdef XGBOOST_USE_CUDF
#include <xgboost/data.h>
#include <xgboost/logging.h>
#include <vector>
#include <xgboost/gpu_column.h>

#include "../common/device_helpers.cuh"
#include "../common/host_device_vector.h"
#include "./cudf.cuh"

namespace xgboost {

using namespace data;

__global__ void unpack_cudf_column_k
  (float* data, size_t n_rows, size_t n_cols, void const* colData, int type) {
  size_t i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i >= n_rows)
    return;
  data[n_cols * i] = ConvertDataElement(colData, i, type);
}

void MetaInfo::SetCUDFInfo(const char* key, std::vector<gpu_column_data *> const& cols,
    int gpu_id) {
  this->SetCUDFInfoImpl(key, cols, gpu_id, false);
}

void MetaInfo::AppendCUDFInfo(const char* key, std::vector<gpu_column_data *> const& cols,
    int gpu_id) {
  this->SetCUDFInfoImpl(key, cols, gpu_id, true);
}

void MetaInfo::SetCUDFInfoImpl(const char* key, std::vector<gpu_column_data *> const& gdfcols,
                               int gpu_id, bool append) {
  size_t n_cols = gdfcols.size();
  CHECK_GT(n_cols, 0);
  size_t n_rows = gdfcols[0]->num_row;
  CHECK_GE(n_rows, 0U);
  for (size_t i = 0; i < n_cols; ++i) {
    CHECK_EQ(gdfcols[i]->null_count, 0) << "all labels and weights must be valid";
    CHECK_EQ(gdfcols[i]->num_row, n_rows) << "all CUDF columns must be of the same size";
  }
  HostDeviceVector<bst_float>* field = nullptr;
  if (!strcmp(key, "label")) {
    field = &labels_;
  } else if (!strcmp(key, "weight")) {
    field = &weights_;
    CHECK_EQ(n_cols, 1) << "only one CUDF column allowed for weights";
  } else {
    LOG(WARNING) << key << ": invalid key value for MetaInfo field";
    return;
  }
  // TODO(canonizer): use the same devices as elsewhere in xgboost
  int device_id = 0;
  GPUSet devices;
  if (gpu_id > 0) {
    device_id = gpu_id;
    devices = GPUSet::All(device_id, 1);
  } else {
    devices = GPUSet::Range(device_id, 1);
  }

  size_t prev_size = (append) ? field->Size() : 0;
  field->Reshard(GPUDistribution::Granular(devices, n_cols));
  field->Resize(prev_size + n_cols * n_rows);
  bst_float* data = field->DevicePointer(device_id);
  data += prev_size;
  for (size_t i = 0; i < n_cols; ++i) {
    int block = 256;
    auto pCol = gdfcols[i];
    unpack_cudf_column_k<<<common::DivRoundUp(n_rows, block), block>>>
      (data + i, n_rows, n_cols, pCol->data_ptr, pCol->type_id);
    dh::safe_cuda(cudaGetLastError());
  }

}
  
}  // namespace xgboost
#endif
