
#include <hip/hip_runtime.h>
/*!
 * Copyright 2018 by xgboost contributors
 */

#ifdef XGBOOST_USE_CUDF

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>

#include <cudf/column/column_view.hpp>

#include "../common/host_device_vector.h"
#include "../common/device_helpers.cuh"

#include "./cudf.cuh"
#include "./simple_csr_source.h"

namespace xgboost {
namespace data {

using cudf::bitmask_type;

struct CsrCudf {
  Entry* data;
  size_t* offsets;
  size_t n_nz;
  size_t n_rows;
  size_t n_cols;
  bst_float missing;
};

void CUDFToCSR(column_view** gdf_data, CsrCudf* csr);

//--- private CUDA functions / kernels
__global__ void cuda_create_csr_k
(void const* cudf_data, bitmask_type const* valid, type_id dtype, int col, Entry* data,
 size_t *offsets, size_t n_rows, bool is_nan_missing, bst_float missing);

__global__ void determine_valid_rec_count_k(void const* cudf_data, type_id dtype, bitmask_type const* valid,
    size_t n_rows,size_t n_cols, size_t *offset, bool is_nan_missing, bst_float missing);

__device__ int WhichBitmap(int record) { return record / 32; }
__device__ int WhichBit(int bit) { return bit % 32; }
__device__ int CheckBit(bitmask_type data, int bit) {

//  gdf_valid_type bit_mask[8] = {1, 2, 4, 8, 16, 32, 64, 128};
//  return data & bit_mask[bit];
  return data & (1U << bit);
}

__device__ bool IsValid(bitmask_type const* valid, int tid) {
  if (valid == nullptr)
    return true;
  int bitmap_idx = WhichBitmap(tid);
  int bit_idx = WhichBit(tid);
  bitmask_type bitmap = valid[bitmap_idx];
  return CheckBit(bitmap, bit_idx);
}

// Convert a CUDF into a CSR CUDF
void CUDFHandleMissingValue(column_view** cudf_data, int n_cols, CsrCudf* csr) {
  // already check its validity
  size_t n_rows = cudf_data[0]->size();

  // the first step is to create an array that counts the number of valid entries per row
  // this is done by each thread looking across its row and checking the valid bits
  int threads = 1024;
  int blocks = (n_rows + threads - 1) / threads;

  size_t* offsets = csr->offsets;
  dh::safe_cuda(cudaMemset(offsets, 0, sizeof(size_t) * (n_rows + 1)));

  if (blocks > 0) {
    for (int i = 0; i < n_cols; ++i) {
      determine_valid_rec_count_k<<<blocks, threads>>>
        (cudf_data[i]->head(), cudf_data[i]->type().id(), cudf_data[i]->null_mask(), n_rows,
            n_cols, offsets, isnan(csr->missing), csr->missing);
      dh::safe_cuda(cudaGetLastError());
      dh::safe_cuda(cudaDeviceSynchronize());
    }
  }

  // compute the number of elements
  thrust::device_ptr<size_t> offsets_begin(offsets);
  int64_t n_elements = thrust::reduce
    (offsets_begin, offsets_begin + n_rows, 0ull, thrust::plus<size_t>());

  // now do an exclusive scan to compute the offsets for where to write data
  thrust::exclusive_scan(offsets_begin, offsets_begin + n_rows + 1, offsets_begin);

  csr->n_rows = n_rows;
  csr->n_cols = n_cols;
  csr->n_nz = n_elements;
}

void CUDFToCSR(column_view** cudf_data, CsrCudf* csr) {
  size_t n_cols = csr->n_cols;
  size_t n_rows = csr->n_rows;
  
  int threads = 256;
  int blocks = (n_rows + threads - 1) / threads;

  // temporary offsets for writing data
  thrust::device_ptr<size_t> offset_begin(csr->offsets);
  thrust::device_vector<size_t> offsets2(offset_begin, offset_begin + n_rows + 1);

  // move the data and create the CSR
  if (blocks > 0) {
    for (int col = 0; col < n_cols; ++col) {
      column_view *cudf = cudf_data[col];
      cuda_create_csr_k<<<blocks, threads>>>
        (cudf->head(), cudf->null_mask(), cudf->type().id(), col, csr->data,
         offsets2.data().get(), n_rows, isnan(csr->missing), csr->missing);
      dh::safe_cuda(cudaGetLastError());
    }
  }
}

// move data over into CSR and possibly convert the format
__global__ void cuda_create_csr_k
(void const* cudf_data, bitmask_type const* valid, type_id dtype, int col,
 Entry* data, size_t* offsets, size_t n_rows, bool is_nan_missing, bst_float missing) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid >= n_rows)
    return;
  size_t offset_idx = offsets[tid];
  if (IsValid(valid, tid)) {
    bst_float v = ConvertDataElement(cudf_data, tid, dtype);
    if (is_nan_missing || v != missing) {
      data[offset_idx].fvalue = v;
      data[offset_idx].index = col;
      ++offsets[tid];
    }
  }
}

// compute the number of valid entries per row
__global__ void determine_valid_rec_count_k(void const* cudf_data, type_id dtype, bitmask_type const* valid,
    size_t n_rows, size_t n_cols, size_t *offset, bool is_nan_missing, bst_float missing) {

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid >= n_rows)
    return;
  if (IsValid(valid, tid)) {
    bst_float v = ConvertDataElement(cudf_data, tid, dtype);
    if (is_nan_missing || v != missing) {
      ++offset[tid];
    }
  }
}

void SimpleCSRSource::InitFromCUDF(column_view** cols, size_t n_cols,
                                   int gpu_id, bst_float missing) {
  CHECK_GT(n_cols, 0);
  int32_t n_rows = cols[0]->size();
  CHECK_GE(n_rows, 0U);
  info.num_col_ = n_cols;
  info.num_row_ = n_rows;

  // TODO(canonizer): use the same devices as by the rest of xgboost
  int device_id = 0;
  GPUSet devices;
  if (gpu_id > 0) {
    device_id = gpu_id;
    devices = GPUSet::All(device_id, 1);
  } else {
    devices = GPUSet::Range(device_id, 1);
  }

  page_.offset.Reshard(GPUDistribution::Overlap(devices, 1));
  page_.offset.Resize(n_rows + 1);

  CsrCudf csr;
  csr.offsets = page_.offset.DevicePointer(device_id);
  csr.n_nz = 0;
  csr.n_rows = n_rows;
  csr.n_cols = n_cols;
  csr.missing = missing;

  CUDFHandleMissingValue(cols, n_cols, &csr);

  // TODO(canonizer): use the real row offsets for the multi-GPU case
  info.num_nonzero_ = csr.n_nz;
  std::vector<size_t> device_offsets{0, csr.n_nz};
  page_.data.Reshard(GPUDistribution::Explicit(devices, device_offsets));
  page_.data.Resize(csr.n_nz);

  csr.data = page_.data.DevicePointer(device_id);
  CUDFToCSR(cols, &csr);

  // Since training copies the data back to the host (as it assumes the dataset
  // is on the host always), move the data from the device to the host. There is
  // no use for the data to sit on the device, if training doesn't use it.
  // Effect this by resharding to an empty device set. This will draw the data
  // from the device to the system memory
  page_.data.Reshard(GPUDistribution());
  page_.offset.Reshard(GPUDistribution());
}

}  // namespace data
}  // namespace xgboost
#endif
