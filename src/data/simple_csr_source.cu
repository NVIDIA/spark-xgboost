
#include <hip/hip_runtime.h>
/*!
 * Copyright 2018 by xgboost contributors
 */

#ifdef XGBOOST_USE_CUDF

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>

#include <xgboost/gpu_column.h>

#include "../common/host_device_vector.h"
#include "../common/device_helpers.cuh"

#include "./cudf.cuh"
#include "./simple_csr_source.h"

namespace xgboost {
namespace data {

struct CsrCudf {
  Entry* data;
  size_t* offsets;
  size_t n_nz;
  size_t n_rows;
  size_t n_cols;
  bst_float missing;
};

void CUDFToCSR(std::vector<gpu_column_data *> const& gdfcols, CsrCudf* csr);

//--- private CUDA functions / kernels
__global__ void cuda_create_csr_k
(void const* cudf_data, uint32_t const* valid, int dtype, int col, Entry* data,
 size_t *offsets, size_t n_rows, bool is_nan_missing, bst_float missing);

__global__ void determine_valid_rec_count_k(void const* cudf_data, int dtype, uint32_t const* valid,
    size_t n_rows,size_t n_cols, size_t *offset, bool is_nan_missing, bst_float missing);

__device__ int WhichBitmap(int record) { return record / 32; }
__device__ int WhichBit(int bit) { return bit % 32; }
__device__ int CheckBit(uint32_t data, int bit) {

//  gdf_valid_type bit_mask[8] = {1, 2, 4, 8, 16, 32, 64, 128};
//  return data & bit_mask[bit];
  return data & (1U << bit);
}

__device__ bool IsValid(uint32_t const* valid, int tid) {
  if (valid == nullptr)
    return true;
  int bitmap_idx = WhichBitmap(tid);
  int bit_idx = WhichBit(tid);
  uint32_t bitmap = valid[bitmap_idx];
  return CheckBit(bitmap, bit_idx);
}

// Convert a CUDF into a CSR CUDF
void CUDFHandleMissingValue(std::vector<gpu_column_data *> const& gdfcols,
    int n_cols, CsrCudf* csr) {
  // already check its validity
  size_t n_rows = gdfcols[0]->num_row;

  // the first step is to create an array that counts the number of valid entries per row
  // this is done by each thread looking across its row and checking the valid bits
  int threads = 1024;
  int blocks = (n_rows + threads - 1) / threads;

  size_t* offsets = csr->offsets;
  dh::safe_cuda(cudaMemset(offsets, 0, sizeof(size_t) * (n_rows + 1)));

  if (blocks > 0) {
    for (int i = 0; i < n_cols; ++i) {
      determine_valid_rec_count_k<<<blocks, threads>>>(
          gdfcols[i]->data_ptr, gdfcols[i]->type_id,
          reinterpret_cast<const unsigned int *>(gdfcols[i]->valid_ptr),
          n_rows, n_cols, offsets, isnan(csr->missing), csr->missing);
      dh::safe_cuda(cudaGetLastError());
      dh::safe_cuda(cudaDeviceSynchronize());
    }
  }

  // compute the number of elements
  thrust::device_ptr<size_t> offsets_begin(offsets);
  int64_t n_elements = thrust::reduce
      (offsets_begin, offsets_begin + n_rows, 0ull, thrust::plus<size_t>());

  // now do an exclusive scan to compute the offsets for where to write data
  thrust::exclusive_scan(offsets_begin, offsets_begin + n_rows + 1, offsets_begin);

  csr->n_rows = n_rows;
  csr->n_cols = n_cols;
  csr->n_nz = n_elements;
}

void CUDFToCSR(std::vector<gpu_column_data *> const& gdfcols, CsrCudf* csr) {
  size_t n_cols = csr->n_cols;
  size_t n_rows = csr->n_rows;

  int threads = 256;
  int blocks = (n_rows + threads - 1) / threads;

  // temporary offsets for writing data
  thrust::device_ptr<size_t> offset_begin(csr->offsets);
  thrust::device_vector<size_t> offsets2(offset_begin, offset_begin + n_rows + 1);

  // move the data and create the CSR
  if (blocks > 0) {
    for (int col = 0; col < n_cols; ++col) {
      gpu_column_data *cudf_column = gdfcols[col];
      cuda_create_csr_k<<<blocks, threads>>>(cudf_column->data_ptr,
          reinterpret_cast<const unsigned int *>(cudf_column->valid_ptr),
          cudf_column->type_id, col, csr->data, offsets2.data().get(), n_rows,
          isnan(csr->missing), csr->missing);
      dh::safe_cuda(cudaGetLastError());
    }
  }
}

// move data over into CSR and possibly convert the format
__global__ void cuda_create_csr_k
(void const* cudf_data, uint32_t const* valid, int dtype, int col,
 Entry* data, size_t* offsets, size_t n_rows, bool is_nan_missing, bst_float missing) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid >= n_rows)
    return;
  size_t offset_idx = offsets[tid];
  if (IsValid(valid, tid)) {
    bst_float v = ConvertDataElement(cudf_data, tid, dtype);
    if (is_nan_missing || v != missing) {
      data[offset_idx].fvalue = v;
      data[offset_idx].index = col;
      ++offsets[tid];
    }
  }
}

// compute the number of valid entries per row
__global__ void determine_valid_rec_count_k(void const* cudf_data, int dtype, uint32_t const* valid,
    size_t n_rows, size_t n_cols, size_t *offset, bool is_nan_missing, bst_float missing) {

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid >= n_rows)
    return;
  if (IsValid(valid, tid)) {
    bst_float v = ConvertDataElement(cudf_data, tid, dtype);
    if (is_nan_missing || v != missing) {
      ++offset[tid];
    }
  }
}

void SimpleCSRSource::InitFromCUDF(std::vector<gpu_column_data *> const& gdfcols,
    int gpu_id, bst_float missing) {
  unsigned int n_cols = gdfcols.size();
  CHECK_GT(n_cols, 0);
  int32_t n_rows = gdfcols[0]->num_row;
  CHECK_GE(n_rows, 0U);
  info.num_col_ = n_cols;
  info.num_row_ = n_rows;

  // TODO(canonizer): use the same devices as by the rest of xgboost
  int device_id = 0;
  GPUSet devices;
  if (gpu_id > 0) {
    device_id = gpu_id;
    devices = GPUSet::All(device_id, 1);
  } else {
    devices = GPUSet::Range(device_id, 1);
  }

  page_.offset.Reshard(GPUDistribution::Overlap(devices, 1));
  page_.offset.Resize(n_rows + 1);

  CsrCudf csr;
  csr.offsets = page_.offset.DevicePointer(device_id);
  csr.n_nz = 0;
  csr.n_rows = n_rows;
  csr.n_cols = n_cols;
  csr.missing = missing;

  CUDFHandleMissingValue(gdfcols, n_cols, &csr);

  // TODO(canonizer): use the real row offsets for the multi-GPU case
  info.num_nonzero_ = csr.n_nz;
  std::vector<size_t> device_offsets{0, csr.n_nz};
  page_.data.Reshard(GPUDistribution::Explicit(devices, device_offsets));
  page_.data.Resize(csr.n_nz);

  csr.data = page_.data.DevicePointer(device_id);
  CUDFToCSR(gdfcols, &csr);

  std::vector<float> tmp(10);

  // Since training copies the data back to the host (as it assumes the dataset
  // is on the host always), move the data from the device to the host. There is
  // no use for the data to sit on the device, if training doesn't use it.
  // Effect this by resharding to an empty device set. This will draw the data
  // from the device to the system memory
  page_.data.Reshard(GPUDistribution());
  page_.offset.Reshard(GPUDistribution());

}

}  // namespace data
}  // namespace xgboost
#endif
